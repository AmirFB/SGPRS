
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
__global__ void k(unsigned long long dt){

  unsigned long long start = clock64();
  while (clock64() < (start+dt));
}

int main(int argc, char *argv[]){

  hipStream_t h, l;
  int hp, lp;
  hipDeviceGetStreamPriorityRange(&lp, &hp);
  std::cout << "lowest priority: " << lp << " highest priority: " << hp << std::endl;
  hipStreamCreateWithPriority(&h, hipStreamDefault, hp);
  hipStreamCreateWithPriority(&l, hipStreamDefault, lp);
  unsigned long long dt = 100000000ULL;
  int blocks = 26*5;
  if (argc > 1) dt *= atoi(argv[1]);
  if (argc > 2) blocks = 1;
  for (int i = 0; i < 5; i++) k<<<blocks, 1024,0,h>>>(dt);
  for (int i = 0; i < 5; i++) k<<<blocks, 1024, 0, l>>>(dt);
  hipDeviceSynchronize();
}